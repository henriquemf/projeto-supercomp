#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <random>
// Importações do thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace std;


struct analisa_configuracao {
    int n_filmes;
    int n_categorias;
    int *start_time;
    int *end_time;
    int *cat_id;
    int *categorias;

    analisa_configuracao(int _n_filmes,
                         int _n_categorias,
                         int* _start_time,
                         int* _end_time,
                         int* _cat_id,
                         int* _categorias
                        ): n_filmes(_n_filmes), 
                           n_categorias(_n_categorias),
                           start_time(_start_time),
                           end_time(_end_time),
                           cat_id(_cat_id),
                           categorias(_categorias)
                        {};

    __device__
    int operator()(const int& x){
        int disponibilidade[24];
        int copy_categorias[10];

        for (int k = 0; k < 24; k++){
            disponibilidade[k] = 0;
        }

        for (int l = 0; l < n_categorias; l++){
            copy_categorias[l] = categorias[l];
        }

        int max_count = 0;

        for (int i = 0; i < n_filmes; i++){
            if (x & (1<<i)){
                if (copy_categorias[cat_id[i] - 1] <= 0) return -1;
                for (int j = start_time[i]; j < end_time[i]; j++){
                    if (disponibilidade[j] == 1) return -1;
                    disponibilidade[j] = 1;
                }
                copy_categorias[cat_id[i] - 1]--;
                max_count++;
            }
        }
        return max_count;
    }
};


int main(){
    int n_filmes, n_categorias;

    cin >> n_filmes >> n_categorias;

    thrust::host_vector<int> categorias(n_categorias);
    thrust::host_vector<int> start_time(n_filmes);
    thrust::host_vector<int> end_time(n_filmes);
    thrust::host_vector<int> cat_id(n_filmes);

    thrust::host_vector<int> result(1);

    for (int i = 0; i < n_categorias; i++){
        cin >> categorias[i];
    }

    int n_start_time, n_end_time;

    for (int i = 0; i < n_filmes; i++){
        cin >> n_start_time >> n_end_time;
        if (n_end_time == 0) n_end_time = 24;
        if (n_start_time < 0) n_start_time = 0;
        if (n_end_time < 0) n_end_time = 0;
        if (n_end_time < n_start_time) n_end_time = 24;
        start_time[i] = n_start_time;
        end_time[i] = n_end_time;
        cin >> cat_id[i];
    }

    cout << "Carregou as entradas" << endl;

    thrust::device_vector<int> device_categorias = categorias;
    thrust::device_vector<int> device_start_time = start_time;
    thrust::device_vector<int> device_end_time = end_time;
    thrust::device_vector<int> device_cat_id = cat_id;

    cout << "Copiou para o device" << endl;

    thrust::device_vector<int> device_configuration(pow(2, n_filmes));
    thrust::sequence(device_configuration.begin(), device_configuration.end());

    cout << "Criou o vetor de configurações" << endl;


    thrust::transform(device_configuration.begin(), device_configuration.end(), device_configuration.begin(), 
                      analisa_configuracao(n_filmes, 
                                           n_categorias,
                                           thrust::raw_pointer_cast(device_start_time.data()), 
                                           thrust::raw_pointer_cast(device_end_time.data()),
                                           thrust::raw_pointer_cast(device_cat_id.data()),
                                           thrust::raw_pointer_cast(device_categorias.data())
                                          )
                     );

    cout << "Aplicou o Transform" << endl;

    thrust::device_vector<int> device_result(1);

    device_result[0] = *thrust::max_element(device_configuration.begin(), device_configuration.end());
    thrust::copy(device_result.begin(), device_result.end(), result.begin());

    cout << "Max Filmes: " << result[0] << endl;

    return 0;
}