#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <random>
// Importações do thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

int N, M;

std::cin >> N >> M;

thrust::host_vector<int> hv_start_times(N+1);
thrust::host_vector<int> hv_end_times(N+1);
thrust::host_vector<int> hv_categories(N+1);
thrust::host_vector<int> hv_L(M+1);

for (int i = 0; i < M; i++) {
    std::cin >> hv_L[i];
}

// Ler os dados do arquivo de entrada
for (int i = 0; i < N; i++) {
    std::cin >> hv_start_times[i] >> hv_end_times[i] >> hv_categories[i];
}
// Carregar os dados do arquivo de entrada na memória da GPU
thrust::device_vector<int> start_times(hv_start_times);
thrust::device_vector<int> end_times(hv_end_times);
thrust::device_vector<int> categories(hv_categories);
thrust::device_vector<int> L(hv_L);


// Criar a matriz de programação dinâmica
thrust::device_vector<int> dp((N+1) * (M+1), 0);

// Inicializar a primeira linha da matriz com zeros
thrust::fill(dp.begin(), dp.begin() + M + 1, 0);

// Preencher a matriz com as soluções para subproblemas menores
for (int i = 1; i <= N; i++) {
    for (int j = 1; j <= M; j++) {
        // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
        int max_count = 0;
        for (int k = 0; k < i; k++) {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= L[j]) {
                max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
            } else {
                max_count = max(max_count, dp[(k*(M+1)) + j]);
            }
        }
        dp[(i*(M+1)) + j] = max_count;
    }
}

// Encontrar o número máximo de filmes que podem ser assistidos
int max_count = 0;
for (int j = 1; j <= M; j++) {
    max_count = max(max_count, dp[(N*(M+1)) + j]);
}

// Escrever o resultado no arquivo de saída
std::cout << max_count << std::endl;